#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <estd/cube.hpp>
#include "../include/ecuda/array.hpp"
#include "../include/ecuda/cube.hpp"

template<typename T>
struct coord_t {
	T x, y, z;
	coord_t( const T& x = T(), const T& y = T(), const T& z = T() ) : x(x), y(y), z(z) {}
	bool operator==( const coord_t& other ) const { return x == other.x and y == other.y and z == other.z; }
	bool operator!=( const coord_t& other ) const { return !operator==(other); }
	friend std::ostream& operator<<( std::ostream& out, const coord_t& coord ) {
		out << "[" << coord.x << "," << coord.y << "," << coord.z << "]";
		return out;
	}
};

typedef coord_t<double> Coordinate;

typedef unsigned char uint8_t;

template<typename T,std::size_t U> __global__
void fetchRow( const ecuda::cube<T> cube, ecuda::array<T,U> array ) {
	T val = *cube.get_allocator().address( cube.data(), 2, 3, cube.get_pitch() );
                //const_pointer np = allocator.address( deviceMemory.get(), columnIndex, depthIndex, pitch );
	printf( "start=[%.05f %.05f %.05f]\n", val.x, val.y, val.z );
	typename ecuda::cube<T>::const_row_type row = cube.get_row( 2, 3 );
	for( typename ecuda::cube<T>::const_row_type::size_type i = 0; i < row.size(); ++i ) array[i] = row[i];
}

int main( int argc, char* argv[] ) {

	estd::cube<Coordinate> hostCube( 3, 4, 5 );
	for( estd::cube<Coordinate>::size_type i = 0; i < hostCube.row_size(); ++i ) {
		for( estd::cube<Coordinate>::size_type j = 0; j < hostCube.column_size(); ++j ) {
			for( estd::cube<Coordinate>::size_type k = 0; k < hostCube.depth_size(); ++k ) {
				hostCube[i][j][k] = Coordinate(i,j,k);
			}
		}
	}

	ecuda::cube<Coordinate> deviceCube( 3, 4, 5 );
	deviceCube << hostCube;

	std::cout << "(1,2,3)=" << hostCube[1][2][3] << std::endl;
	deviceCube >> hostCube;
	std::cout << "(1,2,3)=" << hostCube[1][2][3] << std::endl;

	std::cout << "sizeof(Coordinate)=" << sizeof(Coordinate) << std::endl;

	ecuda::array<Coordinate,3> deviceRow;
	fetchRow<<<1,1>>>( deviceCube, deviceRow );
	CUDA_CHECK_ERRORS();
	CUDA_CALL( hipDeviceSynchronize() );
	std::vector<Coordinate> hostRow;
	deviceRow >> hostRow;
	std::cout << "ROW";
	for( std::vector<Coordinate>::size_type i = 0; i < hostRow.size(); ++i ) std::cout << hostRow[i];
	std::cout << std::endl;

	return EXIT_SUCCESS;

}

