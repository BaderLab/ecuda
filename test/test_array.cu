#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include "../include/ecuda/array.hpp"

__global__ void squareVector( const ecuda::array<float> input, ecuda::array<float> output ) {
	const int index = threadIdx.x;
	output[index] = input[index]*input[index];
}

__global__ void sumVector( const ecuda::array<float> input, ecuda::array<float> output ) {
	const int index = threadIdx.x;
	float sum = 0.0;
	ecuda::array<float>::const_iterator current = input.begin();
	const ecuda::array<float>::const_iterator end = input.end();
	while( current != end ) {
		sum += *current;
		++current;
	}
//	for( ecuda::array<float>::const_iterator iter = input.begin(); iter != input.end(); ++iter ) sum += *iter;
	output[index] = sum;
}

int main( int argc, char* argv[] ) {

	// prepare host vector
	const size_t n = 100;
	std::vector<float> hostVector( n );
	for( size_t i = 0; i < n; ++i ) hostVector[i] = i+1;

	// allocate some device arrays
	ecuda::array<float> deviceArray1( n, 3 ); // should have all 3
	ecuda::array<float> deviceArray2( deviceArray1 ); // should be a copy of deviceArray1
	const ecuda::array<float> deviceArray3( hostVector ); // should be a copy of the host vector

	ecuda::array<float> deviceArray4( n );
	dim3 dimBlock( n, 1 ), dimGrid( 1, 1 );
	squareVector<<<dimGrid,dimBlock>>>( deviceArray3, deviceArray4 );
	CUDA_CHECK_ERRORS
	CUDA_CALL( hipDeviceSynchronize() );

	// copy array to host
	deviceArray4 >> hostVector;
	// print contents
	for( size_t i = 0; i < n; ++i ) std::cout << "test1.hostVector[" << i << "]=" << hostVector[i] << std::endl;

	sumVector<<<dimGrid,dimBlock>>>( deviceArray3, deviceArray4 );
	CUDA_CHECK_ERRORS
	CUDA_CALL( hipDeviceSynchronize() );

	// copy array to host
	deviceArray4 >> hostVector;
	// print contents
	for( size_t i = 0; i < n; ++i ) std::cout << "test2.hostVector[" << i << "]=" << hostVector[i] << std::endl;

	return EXIT_SUCCESS;

}
