#include "hip/hip_runtime.h"
#include <iostream>
#include <list>
#include <vector>

#include "../include/ecuda/algorithm.hpp"
#include "../include/ecuda/allocators.hpp"
#include "../include/ecuda/array.hpp"
#include "../include/ecuda/models.hpp"

#ifndef ECUDA_EMULATE_CUDA_WITH_HOST_ONLY
template<typename T,std::size_t N>
__global__ void testIterators( const ecuda::array<T,N> src, ecuda::array<T,N> dest ) {
	typename ecuda::array<T,N>::iterator result = dest.begin();
	for( typename ecuda::array<T,N>::const_iterator iter = src.begin(); iter != src.end(); ++iter, ++result ) *result = *iter;
}
#endif

int main( int argc, char* argv[] ) {

	std::vector<int> hostVector( 100 );	for( unsigned i = 0; i < 100; ++i ) hostVector[i] = i;

	//ecuda::array<int,100> deviceArray; deviceArray.operator<<( hostVector );
	//if( !ecuda::equal( hostVector.begin(), hostVector.end(), deviceArray.begin() ) ) throw std::runtime_error( "operator<< failed" );

	ecuda::array<int,100> deviceArray;
	ecuda::copy( hostVector.begin(), hostVector.end(), deviceArray.begin() );

	#ifndef ECUDA_EMULATE_CUDA_WITH_HOST_ONLY
	{
		ecuda::array<int,100> deviceArray2;
		testIterators<<<1,1>>>( deviceArray, deviceArray2 );
		CUDA_CHECK_ERRORS();
		CUDA_CALL( hipDeviceSynchronize() );
		std::cout << "EQUAL " << ( deviceArray == deviceArray2 ? "true" : "false" ) << std::endl;
		std::cout << "LESSTHAN " << ( deviceArray < deviceArray2 ? "true" : "false" ) << std::endl;
	}
	#endif

	ecuda::reverse( deviceArray.begin(), deviceArray.end() );

	std::cout << "HOST   VECTOR ="; for( unsigned i = 0; i < hostVector.size(); ++i ) std::cout << " " << hostVector[i]; std::cout << std::endl;
	{
		std::vector<int> tmp( 100 );
		ecuda::copy( deviceArray.begin(), deviceArray.end(), tmp.begin() );
		std::cout << "DEVICE VECTOR ="; for( unsigned i = 0; i < tmp.size(); ++i ) std::cout << " " << tmp[i]; std::cout << std::endl;
	}

	//int* p = 0;
	//typename ecuda::pointer_traits<int*>::unmanaged_pointer q = ecuda::pointer_traits<int*>().make_unmanaged(p);
	//typename ecuda::pointer_traits<int*>::unmanaged_pointer r = ecuda::pointer_traits<int*>::cast_unmanaged(q);

	return EXIT_SUCCESS;

}

