#include "hip/hip_runtime.h"
#include <iostream>
#include <list>
#include <vector>

#include "../include/ecuda/algorithm.hpp"
#include "../include/ecuda/allocators.hpp"
#include "../include/ecuda/matrix.hpp"

template<typename T,typename U>
void testMatrix( const ecuda::model::device_contiguous_row_matrix<T,U>& matrix ) {
	std::vector<T> v( matrix.number_columns() );
	for( unsigned i = 0; i < matrix.number_rows(); ++i ) {
		ecuda::copy( matrix[i].begin(), matrix[i].end(), v.begin() );
		std::cout << "ROW[" << i << "]";
		for( unsigned j = 0; j < matrix.number_columns(); ++j ) std::cout << " " << v[i];
		std::cout << std::endl;
	}
}

template<class InputContainer,class OutputContainer>
inline bool testCopy( const InputContainer& input, OutputContainer& output ) {
	ecuda::fill( output.begin(), output.end(), typename OutputContainer::value_type() );
	ecuda::copy( input.begin(), input.end(), output.begin() );
	return ecuda::equal( input.begin(), input.end(), output.begin() );
}

template<typename T,typename U>
__global__ void testKernel( const ecuda::model::device_contiguous_row_matrix<T,U> matrix, ecuda::model::device_sequence<T,U> vector ) {
	ecuda::copy( matrix.get_column(0).begin(), matrix.get_column(0).end(), vector.begin() );
}

int main( int argc, char* argv[] ) {

	ecuda::model::device_sequence< int, ecuda::shared_ptr<int> > device_sequence_noncontiguous1( ecuda::shared_ptr<int>( ecuda::device_allocator<int>().allocate( 100 ) ), 100 );
	ecuda::model::device_contiguous_sequence< int, ecuda::shared_ptr<int> > device_sequence_contiguous1( ecuda::shared_ptr<int>( ecuda::device_allocator<int>().allocate( 100 ) ), 100 );
	std::vector<int> host_sequence_contiguous1( 100 ); for( std::size_t i = 0; i < host_sequence_contiguous1.size(); ++i ) host_sequence_contiguous1[i] = i;
	std::list<int> host_sequence_noncontiguous1( 100 );

	//if( !testCopy( host_sequence_contiguous1, device_sequence_noncontiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_contiguous1, device_sequence_contiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_contiguous1, host_sequence_contiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	//if( !testCopy( host_sequence_contiguous1, host_sequence_noncontiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );

	//if( !testCopy( host_sequence_noncontiguous1, device_sequence_noncontiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_noncontiguous1, device_sequence_contiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_noncontiguous1, host_sequence_contiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_noncontiguous1, host_sequence_noncontiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );

	//if( !testCopy( device_sequence_noncontiguous1, device_sequence_noncontiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	//if( !testCopy( device_sequence_noncontiguous1, device_sequence_contiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	//if( !testCopy( device_sequence_noncontiguous1, host_sequence_contiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	//if( !testCopy( device_sequence_noncontiguous1, host_sequence_noncontiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );

	//if( !testCopy( device_sequence_contiguous1, device_sequence_noncontiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( device_sequence_contiguous1, device_sequence_contiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( device_sequence_contiguous1, host_sequence_contiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( device_sequence_contiguous1, host_sequence_noncontiguous1 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );

	ecuda::model::device_sequence< double, ecuda::shared_ptr<double> > device_sequence_noncontiguous2( ecuda::shared_ptr<double>( ecuda::device_allocator<double>().allocate( 100 ) ), 100 );
	ecuda::model::device_contiguous_sequence< double, ecuda::shared_ptr<double> > device_sequence_contiguous2( ecuda::shared_ptr<double>( ecuda::device_allocator<double>().allocate( 100 ) ), 100 );
	std::vector<double> host_sequence_contiguous2( 100 );
	std::list<double> host_sequence_noncontiguous2( 100 );

	//if( !testCopy( host_sequence_contiguous1, device_sequence_noncontiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_contiguous1, device_sequence_contiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_contiguous1, host_sequence_contiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_contiguous1, host_sequence_noncontiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );

	//if( !testCopy( host_sequence_noncontiguous1, device_sequence_noncontiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_noncontiguous1, device_sequence_contiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_noncontiguous1, host_sequence_contiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( host_sequence_noncontiguous1, host_sequence_noncontiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );

	//if( !testCopy( device_sequence_noncontiguous1, device_sequence_noncontiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	//if( !testCopy( device_sequence_noncontiguous1, device_sequence_contiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	//if( !testCopy( device_sequence_noncontiguous1, host_sequence_contiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	//if( !testCopy( device_sequence_noncontiguous1, host_sequence_noncontiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );

	//if( !testCopy( device_sequence_contiguous1, device_sequence_noncontiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	//if( !testCopy( device_sequence_contiguous1, device_sequence_contiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( device_sequence_contiguous1, host_sequence_contiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );
	if( !testCopy( device_sequence_contiguous1, host_sequence_noncontiguous2 ) ) throw std::runtime_error( ECUDA_EXCEPTION_MSG("") );

	{
		//std::size_t pitch;
		ecuda::padded_ptr<int> paddedPointer = ecuda::device_pitch_allocator<int>().allocate( 20, 10 );
		//int* p = ecuda::device_pitch_allocator<int>().allocate( 20, 10 );
		//ecuda::padded_ptr<int> paddedPointer( p, pitch, 20, p );
		ecuda::model::device_contiguous_row_matrix< int, ecuda::padded_ptr<int> > matrix( paddedPointer, 10, 20 );
		testMatrix( matrix );
		matrix.get_row(0);
		matrix.get_column(0);
	}


	ecuda::shared_ptr<int> ptr;
	ecuda::model::device_contiguous_row_matrix< int, ecuda::shared_ptr<int> > matrix1( ptr, 10, 20 );
	testMatrix( matrix1 );
	matrix1.get_row(0);
	matrix1.get_column(0);

	ecuda::model::device_contiguous_row_matrix< int, int* > matrix2( ptr.get(), 10, 20 );
	testMatrix( matrix2 );
	matrix2.get_row(0);
	matrix2.get_column(0);

	//ecuda::model::device_contiguous_row_matrix< int, ecuda::padded_ptr< int, ecuda::shared_ptr<int> > > matrix3( ecuda::padded_ptr< int, ecuda::shared_ptr<int> >( ecuda::shared_ptr<int>(), 0 ) );
	//testMatrix( matrix3 );
	//matrix3.get_row(0);
	//matrix3.get_column(0);
	//testCopy( matrix3, matrix3 );

	testKernel<<<1,1>>>( matrix1, device_sequence_noncontiguous1 );

	return EXIT_SUCCESS;

}

