#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include "../include/ecuda/vector.hpp"

__global__ void testKernel( const ecuda::vector<float> input, ecuda::vector<float> output )
//__global__ void testKernel( const ecuda::array<float>::DevicePayload in, ecuda::array<float>::DevicePayload out )
{
	const int index = threadIdx.x;
	//const ecuda::array<float> input( in );
	//ecuda::array<float> output( out );
	//printf( "index=%i value_before=%.2f\n", index, input[index] );
	for( ecuda::vector<float>::const_iterator iter = input.begin(); iter != input.end(); ++iter ) {
		output.front() = *iter;
	}
	output[index] = input[index]*static_cast<float>(index);
        printf( "index=%i value_before=%.2f value_after=%.2f\n", index, input[index], output[index] );
	//printf( "value_after=%.2f\n", output[index] );
}


int main( int argc, char* argv[] ) {
std::cerr << "step1" << std::endl;
	std::vector<float> hostVectorInput( 100 );
std::cerr << "step2" << std::endl;
	for( size_t i = 0; i < 100; ++i ) hostVectorInput[i] = i+1;
	//ecuda::array<float> deviceVectorInput( &hostVectorInput.front(), hostVectorInput.size() );
std::cerr << "step3" << std::endl;
	ecuda::vector<float> deviceVectorInput( hostVectorInput );
std::cerr << "step4" << std::endl;
	ecuda::vector<float> deviceVectorOutput( hostVectorInput.size() );
std::cerr << "step5" << std::endl;

	dim3 dimBlock( 100, 1 ), dimGrid( 1, 1 );
	//testKernel<<<dimGrid,dimBlock>>>( deviceVectorInput.passToDevice(), deviceVectorOutput.passToDevice() );
std::cerr << "step6" << std::endl;
	testKernel<<<dimGrid,dimBlock>>>( deviceVectorInput, deviceVectorOutput );
std::cerr << "step7" << std::endl;
	CUDA_CHECK_ERRORS
std::cerr << "step8" << std::endl;
	CUDA_CALL( hipDeviceSynchronize() );
std::cerr << "COMPLETE" << std::endl;

	std::vector<float> hostVectorOutput;
std::cerr << "step9" << std::endl;
	deviceVectorOutput >> hostVectorOutput;
std::cerr << "step10" << std::endl;
	for( size_t i = 0; i < hostVectorOutput.size(); ++i ) std::cout << "[" << i << "]=" << hostVectorOutput[i] << std::endl;

	return EXIT_SUCCESS;

}
