#include "hip/hip_runtime.h"
//#define NDEBUG
//#include <cassert>

#include <iostream>
#include <cstdio>
#include <vector>
#include <estd/matrix.hpp>
#include "../include/ecuda/array.hpp"
#include "../include/ecuda/matrix.hpp"

template<typename T>
struct coord_t {
	T x, y;
	coord_t( const T& x = T(), const T& y = T() ) : x(x), y(y) {}
	bool operator==( const coord_t& other ) const { return x == other.x and y == other.y; }
	bool operator!=( const coord_t& other ) const { return !operator==(other); }
	friend std::ostream& operator<<( std::ostream& out, const coord_t& coord ) {
		out << "[" << coord.x << "," << coord.y << "]";
		return out;
	}
};

typedef coord_t<double> Coordinate;

typedef unsigned char uint8_t;

template<typename T> __global__
void kernel_checkMatrixProperties(
	const ecuda::matrix<T> constMatrix,
	ecuda::matrix<T> matrix,
	ecuda::vector<int> empties,
	ecuda::vector<typename ecuda::matrix<T>::size_type> sizes,
	ecuda::vector<typename ecuda::matrix<T>::pointer> pointers,
	ecuda::vector<typename ecuda::matrix<T>::const_pointer> constPointers
)
{
	const int row = blockIdx.x;
	const int column = threadIdx.x;
	if( row < matrix.number_rows() and column < matrix.number_columns() ) {
		const int index = row*matrix.number_columns()+column;
		empties[index] = constMatrix.empty() ? 1 : 0;
		sizes[index] = constMatrix.size();
		pointers[index] = matrix.data();
		constPointers[index] = constMatrix.data();
	}
}

template<typename T> __global__
void kernel_checkMatrixAccessors(
	const ecuda::matrix<T> srcMatrix,
	ecuda::matrix<T> srcMatrixNonConst,
	ecuda::matrix<T> destMatrix,
	ecuda::vector<T> srcFronts,
	ecuda::vector<T> srcBacks,
	ecuda::vector<T> srcFrontsNonConst,
	ecuda::vector<T> srcBacksNonConst
)
{
	const int row = blockIdx.x;
	const int column = threadIdx.x;
	if( row < srcMatrix.number_rows() and column < srcMatrix.number_columns() ) {
		const int index = row*srcMatrix.number_columns()+column;
		destMatrix[row][column] = srcMatrix[row][column];
		srcFronts[index] = srcMatrix.front();
		srcBacks[index] = srcMatrix.back();
		srcFrontsNonConst[index] = srcMatrixNonConst.front();
		srcBacksNonConst[index] = srcMatrixNonConst.back();
	}
}


template<typename T> __global__
void kernel_checkDeviceIterators(
	const ecuda::matrix<T> srcMatrix,
	ecuda::matrix<T> destMatrix
)
{
	typename ecuda::matrix<T>::const_iterator srcIterator = srcMatrix.begin();
	typename ecuda::matrix<T>::iterator destIterator = destMatrix.begin();
	for( ; srcIterator != srcMatrix.end() and destIterator != destMatrix.end(); ++srcIterator, ++destIterator ) *destIterator = *srcIterator;
}


int main( int argc, char* argv[] ) {

	std::cout << "Testing ecuda::matrix..." << std::endl;

	std::vector<int> testResults;

	// Test 1: default constructor, copy to host and general info
	std::cerr << "Test 1" << std::endl;
	{
		bool passed = true;
		{
			ecuda::matrix<int> deviceMatrix;
			if( deviceMatrix.size() ) passed = false;
			if( !deviceMatrix.empty() ) passed = false;
			if( deviceMatrix.number_rows() ) passed = false;
			if( deviceMatrix.number_columns() ) passed = false;
		}
		{
			const ecuda::matrix<int> deviceMatrix( 10, 20 );
			if( deviceMatrix.size() != 200 ) passed = false;
			if( deviceMatrix.empty() ) passed = false;
			if( !deviceMatrix.data() ) passed = false;
			std::vector<int> hostVector;
			deviceMatrix >> hostVector;
			if( hostVector.size() != 200 ) passed = false;
			for( std::vector<int>::size_type i = 0; i < hostVector.size(); ++i ) if( hostVector[i] ) passed = false;
		}
		ecuda::matrix<int> deviceMatrix( 10, 20, 3 );
		if( deviceMatrix.size() != 200 ) passed = false;
		if( deviceMatrix.empty() ) passed = false;
		if( !deviceMatrix.data() ) passed = false;
		std::vector<int> hostVector;
		deviceMatrix >> hostVector;
		if( hostVector.size() != 200 ) passed = false;
		for( std::vector<int>::size_type i = 0; i < hostVector.size(); ++i ) if( hostVector[i] != 3 ) passed = false;
		testResults.push_back( passed ? 1 : 0 );
	}

	// Test 2: information is correct on device
	std::cerr << "Test 2" << std::endl;
	{
		std::vector<Coordinate> hostVector( 200 );
		std::vector<Coordinate>::size_type index = 0;
		for( unsigned i = 0; i < 10; ++i )
			for( unsigned j = 0; j < 20; ++j, ++index )
				hostVector[i] = Coordinate(i,j);
		ecuda::matrix<Coordinate> deviceMatrix( 10, 20 );
		deviceMatrix.assign( hostVector.begin(), hostVector.end() );
		ecuda::vector<int> deviceEmpties( 200, -1 );
		ecuda::vector<ecuda::matrix<Coordinate>::size_type> deviceSizes( 200 );
		ecuda::vector<ecuda::matrix<Coordinate>::pointer> devicePointers( 200 );
		ecuda::vector<ecuda::matrix<Coordinate>::const_pointer> deviceConstPointers( 200 );
		kernel_checkMatrixProperties<<<10,20>>>( deviceMatrix, deviceMatrix, deviceEmpties, deviceSizes, devicePointers, deviceConstPointers );
		CUDA_CHECK_ERRORS();
		CUDA_CALL( hipDeviceSynchronize() );
		std::vector<int> hostEmpties( 200, -1 );
		std::vector<ecuda::matrix<Coordinate>::size_type> hostSizes( 200 );
		std::vector<ecuda::matrix<Coordinate>::pointer> hostPointers( 200 );
		std::vector<ecuda::matrix<Coordinate>::const_pointer> hostConstPointers( 200 );
		deviceEmpties >> hostEmpties;
		deviceSizes >> hostSizes;
		devicePointers >> hostPointers;
		deviceConstPointers >> hostConstPointers;
		bool passed = true;
		for( std::vector<int>::size_type i = 0; i < hostEmpties.size(); ++i ) if( hostEmpties[i] != 0 ) passed = false;
		for( std::vector<ecuda::vector<int>::size_type>::size_type i = 0; i < hostSizes.size(); ++i ) if( hostSizes[i] != 200 ) passed = false;
		for( std::vector<ecuda::vector<int>::pointer>::size_type i = 0; i < hostPointers.size(); ++i ) if( hostPointers[i] != deviceMatrix.data() ) passed = false;
		for( std::vector<ecuda::vector<int>::const_pointer>::size_type i = 0; i < hostConstPointers.size(); ++i ) if( hostConstPointers[i] != deviceMatrix.data() ) passed = false;
		testResults.push_back( passed ? 1 : 0 );
	}

	// Test 3: C++11 assignment
	#ifdef __CPP11_SUPPORTED__
	{
		std::cerr << "Test 3" << std::endl;
		ecuda::matrix<Coordinate> deviceMatrix( 2, 2 );
		deviceMatrix.assign( { Coordinate(0,0), Coordinate(0,1), Coordinate(1,0), Coordinate(1,1) } );
		std::vector<Coordinate> hostVector;
		deviceMatrix >> hostVector;
		bool passed = true;
		std::vector<Coordinate>::size_type index = 0;
		for( std::vector<Coordinate>::size_type i = 0; i < 2; ++i ) {
			for( std::vector<Coordinate>::size_type j = 0; j < 2; ++j, ++index ) {
				if( hostVector[index] != Coordinate(i,j) ) passed = false;
			}
		}
		testResults.push_back( passed ? 1 : 0 );
	}
	#else
	std::cerr << "Test 3 (skipped)" << std::endl;
	testResults.push_back( -1 );
	#endif

	// Test 4: index accessors, front(), and back()
	std::cerr << "Test 4" << std::endl;
	{
		std::vector<Coordinate> hostVector( 10*20 );
		unsigned index = 0;
		for( unsigned i = 0; i < 10; ++i ) {
			for( unsigned j = 0; j < 20; ++j, ++index ) {
				hostVector[index] = Coordinate(i,j);
			}
		}
		ecuda::matrix<Coordinate> deviceMatrix( 10, 20 );
		deviceMatrix.assign( hostVector.begin(), hostVector.end() );
		ecuda::matrix<Coordinate> destDeviceMatrix( 10, 20 );
		ecuda::vector<Coordinate> deviceFronts( 10*20, -1 );
		ecuda::vector<Coordinate> deviceBacks( 10*20, -1 );
		ecuda::vector<Coordinate> deviceFrontsNonConst( 10*20, -1 );
		ecuda::vector<Coordinate> deviceBacksNonConst( 10*20, -1 );
		kernel_checkMatrixAccessors<<<10,20>>>( deviceMatrix, deviceMatrix, destDeviceMatrix, deviceFronts, deviceBacks, deviceFrontsNonConst, deviceBacksNonConst );
		CUDA_CHECK_ERRORS();
		CUDA_CALL( hipDeviceSynchronize() );

		bool passed = true;
		std::vector<Coordinate> hostResults;

		destDeviceMatrix >> hostResults;

		for( std::vector<Coordinate>::size_type i = 0; i < hostResults.size(); ++i ) if( hostResults[i] != Coordinate(i/20,i%20) ) passed = false;

		deviceFronts >> hostResults;
		for( std::vector<Coordinate>::size_type i = 0; i < hostResults.size(); ++i ) if( hostResults[i] != Coordinate(0,0) ) passed = false;

		deviceBacks >> hostResults;
		for( std::vector<Coordinate>::size_type i = 0; i < hostResults.size(); ++i ) if( hostResults[i] != Coordinate(9,19) ) passed = false;

		deviceFrontsNonConst >> hostResults;
		for( std::vector<Coordinate>::size_type i = 0; i < hostResults.size(); ++i ) if( hostResults[i] != Coordinate(0,0) ) passed = false;

		deviceBacksNonConst >> hostResults;
		for( std::vector<Coordinate>::size_type i = 0; i < hostResults.size(); ++i ) if( hostResults[i] != Coordinate(9,19) ) passed = false;

		testResults.push_back( passed ? 1 : 0 );

	}

	// Test 5: check device iterators
	{
		std::vector<Coordinate> hostVector( 10*20 );
		unsigned index = 0;
		for( unsigned i = 0; i < 10; ++i ) {
			for( unsigned j = 0; j < 20; ++j, ++index ) {
				hostVector[index] = Coordinate(i,j);
			}
		}
		ecuda::matrix<Coordinate> srcDeviceMatrix( 10, 20 );
		srcDeviceMatrix.assign( hostVector.begin(), hostVector.end() );
		ecuda::matrix<Coordinate> destDeviceMatrix( 10, 20 );
		kernel_checkDeviceIterators<<<1,1>>>( srcDeviceMatrix, destDeviceMatrix );
		CUDA_CHECK_ERRORS();
		CUDA_CALL( hipDeviceSynchronize() );
		std::fill( hostVector.begin(), hostVector.end(), Coordinate(9000,9000) );
		destDeviceMatrix >> hostVector;
		bool passed = true;
		for( std::vector<Coordinate>::size_type i = 0; i < hostVector.size(); ++i ) if( hostVector[i] != Coordinate(i/20,i%20) ) passed = false;
		testResults.push_back( passed ? 1 : 0 );
	}

	for( std::vector<bool>::size_type i = 0; i < testResults.size(); ++i ) std::cout << ( testResults[i] == 1 ? "P" : ( testResults[i] == -1 ? "?" : "F" ) ) << "|";

	return EXIT_SUCCESS;

}
