#include "hip/hip_runtime.h"
#include <iostream>
#include <list>
//#include <initializer_list>
#include <vector>

#include "../include/ecuda/algorithm.hpp"
#include "../include/ecuda/allocators.hpp"
#include "../include/ecuda/array.hpp"
#include "../include/ecuda/matrix.hpp"
#include "../include/ecuda/models.hpp"
#include "../include/ecuda/vector.hpp"

template<typename T>
__global__ void testIterators( const typename ecuda::matrix<T>::kernel src, typename ecuda::matrix<T>::kernel dest ) {
	typename ecuda::matrix<T>::iterator result = dest.begin();
	//typename ecuda::matrix<T>::const_iterator result2 = result;
	for( typename ecuda::matrix<T>::const_iterator iter = src.begin(); iter != src.end(); ++iter, ++result ) *result = *iter;
}

template<typename T>
__global__ void testIterators2( const ecuda::matrix<T> src, ecuda::matrix<T> dest ) {
	for( typename ecuda::matrix<T>::size_type i = 0; i < src.number_columns(); ++i ) {
		typename ecuda::matrix<T>::const_column_type srcColumn = src.get_column(i);
		typename ecuda::matrix<T>::column_type destColumn = dest.get_column(i);
		ecuda::copy( srcColumn.begin(), srcColumn.end(), destColumn.begin() );
	}
}

int main( int argc, char* argv[] ) {

	std::vector<int> hostVector( 100 );
	for( unsigned i = 0; i < 100; ++i ) hostVector[i] = i;

	ecuda::matrix<int> deviceMatrix( 5, 20 );
	// below needs to be made to work
	ecuda::copy( hostVector.begin(), hostVector.end(), deviceMatrix.begin() );
	{
		ecuda::matrix<int> deviceMatrix2( 5, 20 );
		testIterators2<<<1,1>>>( deviceMatrix, deviceMatrix2 );
		CUDA_CHECK_ERRORS();
		CUDA_CALL( hipDeviceSynchronize() );

		// need alternative to below
		//ecuda::copy( deviceMatrix.get_column(0).begin(), deviceMatrix.get_column(0).end(), deviceMatrix2.get_column(1).begin() );

		std::cout << "EQUAL " << ( deviceMatrix == deviceMatrix2 ? "true" : "false" ) << std::endl;
		std::cout << "LESS THAN " << ( deviceMatrix < deviceMatrix2 ? "true" : "false" ) << std::endl;
	}

	ecuda::matrix_transpose( deviceMatrix );

	{
		//ecuda::matrix<int> deviceMatrix2( 2, 2 );
		//deviceMatrix2.assign( { 1, 2, 3, 4 } );
	}

	return EXIT_SUCCESS;

}

