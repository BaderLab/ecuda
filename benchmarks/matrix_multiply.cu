#include "hip/hip_runtime.h"
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>
#include "../include/ecuda/matrix.hpp"
#include "../include/ecuda/event.hpp"

#define PRINT_MATRIX

template<typename T>
__global__ void matrixMultiply(
	const T* A,
	std::size_t pitchA,
	const T* B,
	std::size_t pitchB,
	const std::size_t n, const std::size_t m, const std::size_t p,
	T* AB,
	const std::size_t pitchAB
);
template<typename T> __global__ void matrixMultiply( const ecuda::matrix<T> A, const ecuda::matrix<T> B, ecuda::matrix<T> AB );


float cpuMatrixMultiply( const std::size_t n, const std::size_t m, const std::size_t p, const double* pool );
float cudaMatrixMultiply( const int numThreads, const std::size_t n, const std::size_t m, const std::size_t p, const double* pool );
float ecudaMatrixMultiply( const int numThreads, const std::size_t n, const std::size_t m, const std::size_t p, const double* pool );

int main( int argc, char* argv[] ) {

	const std::size_t THREADS = 480;
	const std::size_t n = 1000;
	const std::size_t m = 1000;
	const std::size_t p = 1000;

	std::vector<double> pool( n*m + m*p );
	for( std::vector<double>::iterator iter = pool.begin(); iter != pool.end(); ++iter ) *iter = static_cast<double>(rand()) / static_cast<double>(RAND_MAX);

	//std::cout << "MATRIX MULTIPLICATION CPU  : " << std::fixed <<   cpuMatrixMultiply(          n, m, p, &pool.front() ) << " ms" << std::endl;
	std::cout << "MATRIX MULTIPLICATION CUDA : " << std::fixed <<  cudaMatrixMultiply( THREADS, n, m, p, &pool.front() ) << " ms" << std::endl;
	std::cout << "MATRIX MULTIPLICATION ECUDA: " << std::fixed << ecudaMatrixMultiply( THREADS, n, m, p, &pool.front() ) << " ms" << std::endl;

	return EXIT_SUCCESS;

}

template<typename T>
__global__ void matrixMultiply(	const T* A,	std::size_t pitchA,	const T* B,	std::size_t pitchB,	const std::size_t n, const std::size_t m, const std::size_t p, T* AB, const std::size_t pitchAB ) {
	const int x = blockIdx.x*blockDim.x+threadIdx.x; // row
	const int y = blockIdx.y*blockDim.y+threadIdx.y; // column
	if( x < n and y < p ) {
		T result = 0;
		for( std::size_t i = 0; i < m; ++i ) {
			const T A_ik = *(reinterpret_cast<const T*>( reinterpret_cast<const char*>(A)+(pitchA*x) )+i);
			const T B_kj = *(reinterpret_cast<const T*>( reinterpret_cast<const char*>(B)+(pitchB*i) )+y);
			result += A_ik * B_kj;
		}
		*reinterpret_cast<T*>( reinterpret_cast<char*>(AB)+(pitchAB*y+x*sizeof(T)) ) = result;
	}
}

template<typename T>
__global__ void matrixMultiply(	const ecuda::matrix<T> A, const ecuda::matrix<T> B,	ecuda::matrix<T> AB ) {
	const int x = blockIdx.x*blockDim.x+threadIdx.x; // row
	const int y = blockIdx.y*blockDim.y+threadIdx.y; // column
	//const ecuda::matrix<A>::size_type n = A.number_rows();
	//const ecuda::matrix<A>::size_type m = A.number_columns();
	//const ecuda::matrix<B>::size_type p = B.number_rows();
	//if( x < n and y < p ) {
	if( x < A.number_rows() and y < B.number_columns() ) {
		T result = 0;
		for( std::size_t i = 0; i < A.number_columns(); ++i ) result += A.at(x,i) * B.at(i,y); //[x][i] * B[i][y];
		AB.at( x, y ) = result;
		//AB[x][y] = result;
	}
}

float cudaMatrixMultiply( const int numThreads, const std::size_t n, const std::size_t m, const std::size_t p, const double* pool ) {

	ecuda::event start, stop;

	double *A, *B, *AB;
	std::size_t pitchA, pitchB, pitchAB;

	hipMallocPitch( &A, &pitchA, m*sizeof(double), n );
	hipMallocPitch( &B, &pitchB, p*sizeof(double), m );
	hipMallocPitch( &AB, &pitchAB, p*sizeof(double), n );

	hipMemcpy2D( A, pitchA, pool, sizeof(double)*m, sizeof(double)*m, n, hipMemcpyHostToDevice );
	hipMemcpy2D( B, pitchB, pool+(m*n), sizeof(double)*p, sizeof(double)*p, m, hipMemcpyHostToDevice );
	hipMemset( AB, 0, n*p*sizeof(double) );

	dim3 grid( n, (p+numThreads-1)/numThreads ), threads( 1, numThreads );
	start.record();
	matrixMultiply<<<grid,threads>>>( A, pitchA, B, pitchB, n, m, p, AB, pitchAB );
	stop.record();

	CUDA_CHECK_ERRORS();
	stop.synchronize();

	#ifdef PRINT_MATRIX
	std::vector<double> hostVector( 10*10 );
	for( std::size_t i = 0; i < 10; ++i ) {
		hipMemcpy( &hostVector[i*10], reinterpret_cast<char*>(AB)+(pitchAB*i), sizeof(double)*10, hipMemcpyDeviceToHost );
	}
	for( std::size_t i = 0; i < 10; ++i ) {
		std::cout << "ROW[" << i << "]";
		for( std::size_t j = 0; j < 10; ++j ) std::cout << " " << std::fixed << hostVector[i*10+j];
		std::cout << std::endl;
	}
	#endif

	hipFree( A );
	hipFree( B );
	hipFree( AB );

	return ( stop - start );

}

float ecudaMatrixMultiply( const int numThreads, const std::size_t n, const std::size_t m, const std::size_t p, const double* pool ) {

	ecuda::event start, stop;

	ecuda::matrix<double> A( n, m );
	ecuda::matrix<double> B( m, p );
	ecuda::matrix<double> AB( n, p );

	ecuda::host_array_proxy<const double> randomNumbers( pool, n*m+m*p );

	A.assign( randomNumbers.begin(), randomNumbers.begin()+(n*m) );
	B.assign( randomNumbers.begin()+(n*m), randomNumbers.end() );
	AB.fill( 0.0 );

	//hipMemcpy2D( A.data(), A.get_pitch(), pool, sizeof(double)*m, sizeof(double)*m, n, hipMemcpyHostToDevice );
	//hipMemcpy2D( B.data(), B.get_pitch(), pool+(m*n), sizeof(double)*p, sizeof(double)*p, m, hipMemcpyHostToDevice );
	//hipMemset( AB.data(), 0, n*p*sizeof(double) );

	dim3 grid( n, (p+numThreads-1)/numThreads ), threads( 1, numThreads );
	start.record();
	matrixMultiply<<<grid,threads>>>( A, B, AB );
	stop.record();

	CUDA_CHECK_ERRORS();
	stop.synchronize();

	#ifdef PRINT_MATRIX
	std::vector<double> hostVector( 10*10 );
	for( std::size_t i = 0; i < 10; ++i ) {
		std::vector<double> hostRow( AB.number_columns() );
		AB[i] >> hostRow;
		for( std::size_t j = 0; j < 10; ++j ) hostVector[i*10+j] = hostRow[j];
	}
	for( std::size_t i = 0; i < 10; ++i ) {
		std::cout << "ROW[" << i << "]";
		for( std::size_t j = 0; j < 10; ++j ) std::cout << " " << std::fixed << hostVector[i*10+j];
		std::cout << std::endl;
	}
	#endif

	return ( stop - start );

}

float cpuMatrixMultiply( const std::size_t n, const std::size_t m, const std::size_t p, const double* pool ) {

	ecuda::event start, stop;
	start.record();

	std::vector<double> A( n*m );
	std::vector<double> B( m*p );
	std::vector<double> AB( n*p );

	for( std::vector<double>::size_type i = 0; i < A.size(); ++i ) A[i] = *(pool+i);
	for( std::vector<double>::size_type i = 0; i < B.size(); ++i ) B[i] = *(pool+(n*m+i));

	for( std::size_t i = 0; i < n; ++i ) {
		for( std::size_t j = 0; j < p; ++j ) {
			double sum = 0.0;
			for( std::size_t k = 0; k < m; ++k ) sum += A[i*m+k] * B[k*p+j];
			AB[i*p+j] = sum;
		}
	}

	stop.record();

	//for( std::size_t i = 0; i < n; ++i ) {
	//	std::cout << "ROW[" << i << "]";
	//	for( std::size_t j = 0; j < p; ++j ) std::cout << " " << std::fixed << AB[i*p+j];
	//	std::cout << std::endl;
	//}

	return ( stop - start );

}
