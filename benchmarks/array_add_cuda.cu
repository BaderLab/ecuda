#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "../include/ecuda/event.hpp"

#include "config.hpp"
#ifndef BENCHMARK_THREADS
#define BENCHMARK_THREADS 480
#endif

typedef double value_type;

template<typename T,std::size_t N>
__global__
void copyArray( const T* src, T* dest )
{
	const int t = blockIdx.x*blockDim.x+threadIdx.x;
	if( t < N ) dest[t] = src[t];
}

int main( int argc, char* argv[] )
{

	const std::size_t N = 1000000;

	ecuda::event start, stop;
	start.record();

	std::vector<value_type> hostSequence1( N );
	{
		std::size_t n = 0;
		for( typename std::vector<value_type>::iterator iter = hostSequence1.begin(); iter != hostSequence1.end(); ++iter, ++n ) *iter = static_cast<value_type>(n);
	}

	value_type *deviceSequence1, *deviceSequence2;
	CUDA_CALL( hipMalloc( &deviceSequence1, N*sizeof(value_type) ) );
	CUDA_CALL( hipMalloc( &deviceSequence2, N*sizeof(value_type) ) );

	CUDA_CALL( hipMemcpy( deviceSequence1, &hostSequence1.front(), N*sizeof(value_type), hipMemcpyHostToDevice ) );

	dim3 grid( (N+BENCHMARK_THREADS-1)/BENCHMARK_THREADS ), threads( BENCHMARK_THREADS );
	CUDA_CALL_KERNEL_AND_WAIT( copyArray<value_type,N><<<grid,threads>>>( deviceSequence1, deviceSequence2 ) );

	value_type *hostSequence2;
	CUDA_CALL( hipHostMalloc( &hostSequence2, N*sizeof(value_type), hipHostMallocDefault ) );

	CUDA_CALL( hipMemcpy( hostSequence2, deviceSequence2, N*sizeof(value_type), hipMemcpyDeviceToHost ) );

	const bool isEqual = std::equal( hostSequence2, hostSequence2+N, hostSequence1.begin() );

	hipHostFree( hostSequence2 );
	hipFree( deviceSequence1 );
	hipFree( deviceSequence2 );

	stop.record();
	stop.synchronize();

	if( isEqual )
		std::cout << "Test successful." << std::endl;
	else
		std::cout << "Test failed." << std::endl;

	std::cout << "Execution Time: " << (stop-start) << "ms" << std::endl;

	return EXIT_SUCCESS;

}

