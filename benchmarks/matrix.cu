#include "hip/hip_runtime.h"
#include <ctime>
#include <iomanip>
#include <iostream>
#include <vector>
//#include <estd/matrix.hpp>
#include "../include/ecuda/matrix.hpp"
#include "../include/ecuda/event.hpp"

template<typename T>
__global__ void matrixMultiply(
	const T* A,
	std::size_t pitchA,
	const T* B,
	std::size_t pitchB,
	std::size_t n, std::size_t m, std::size_t p,
	T* AB,
	std::size_t pitchAB
);

template<typename T> __global__ void matrixMultiply( const ecuda::matrix<T> A, const ecuda::matrix<T> B, ecuda::matrix<T> AB );

float cudaMatrixMultiply( const int numThreads, const std::size_t n = 100, const std::size_t m = 100, const std::size_t p = 100 );
float ecudaMatrixMultiply( const int numThreads, const std::size_t n = 100, const std::size_t m = 100, const std::size_t p = 100 );

int main( int argc, char* argv[] ) {

	const std::size_t THREADS = 480;
	const std::size_t n = 1000;
	const std::size_t m = 1000;
	const std::size_t p = 1000;

	std::cout << "STANDARD: " << std::fixed <<  cudaMatrixMultiply( THREADS, n, m, p ) << " ms" << std::endl;
	std::cout << "ECUDA   : " << std::fixed << ecudaMatrixMultiply( THREADS, n, m, p ) << " ms" << std::endl;

	return EXIT_SUCCESS;

}

template<typename T>
__global__ void matrixMultiply(
	const T* A,
	std::size_t pitchA,
	const T* B,
	std::size_t pitchB,
	std::size_t n, std::size_t m, std::size_t p,
	T* AB,
	std::size_t pitchAB
)
{
	const int x = blockIdx.x*blockDim.x+threadIdx.x; // row
	const int y = blockIdx.y*blockDim.y+threadIdx.y; // column
	if( x < n and y < p ) {
		T result = 0;
		for( std::size_t i = 0; i < m; ++i ) {
			const T A_ik = *(reinterpret_cast<const T*>( reinterpret_cast<const char*>(A)+(pitchA*i) )+x);
			const T B_kj = *(reinterpret_cast<const T*>( reinterpret_cast<const char*>(B)+(pitchB*y) )+i);
			result += A_ik * B_kj;
		}
		*(reinterpret_cast<T*>( reinterpret_cast<char*>(AB)+(pitchAB*x) )+y) = result;
	}
}

template<typename T>
__global__ void matrixMultiply(
	const ecuda::matrix<T> A,
	const ecuda::matrix<T> B,
	ecuda::matrix<T> AB
)
{
	const int x = blockIdx.x*blockDim.x+threadIdx.x; // row
	const int y = blockIdx.y*blockDim.y+threadIdx.y; // column
	if( x < A.number_rows() and y < B.number_columns() ) {
		T result = 0;
		for( std::size_t i = 0; i < A.number_columns(); ++i ) result += A[x][i] * B[i][y];
		AB[x][y] = result;
	}
}

float cudaMatrixMultiply( const int numThreads, const std::size_t n, const std::size_t m, const std::size_t p ) {

	ecuda::event start, stop;
	start.record();

	double *A, *B, *AB;
	std::size_t pitchA, pitchB, pitchAB;

	hipMallocPitch( &A, &pitchA, n, m );
	hipMallocPitch( &B, &pitchB, m, p );
	hipMallocPitch( &AB, &pitchAB, n, p );

	dim3 grid( n, (p+numThreads-1)/numThreads ), threads( 1, numThreads );
	matrixMultiply<<<grid,threads>>>( A, pitchA, B, pitchB, n, m, p, AB, pitchAB );

	hipFree( A );
	hipFree( B );
	hipFree( AB );

	stop.record();
	stop.synchronize();

	return ( stop - start );

}

float ecudaMatrixMultiply( const int numThreads, const std::size_t n, const std::size_t m, const std::size_t p ) {

	ecuda::event start, stop;
	start.record();

	ecuda::matrix<double> A( n, m );
	ecuda::matrix<double> B( m, p );
	ecuda::matrix<double> AB( n, p );

	dim3 grid( n, (p+numThreads-1)/numThreads ), threads( 1, numThreads );
	matrixMultiply<<<grid,threads>>>( A, B, AB );

	stop.record();
	stop.synchronize();

	return ( stop - start );

}
