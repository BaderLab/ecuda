#include "hip/hip_runtime.h"
#include <ctime>
#include <vector>
#include "../include/ecuda/array.hpp"

template<typename T>
__global__
void squareArray( ecuda::array<T> input ) {
	const int index = blockIdx.x*blockDim.x+threadIdx.x;
	if( index < input.size() ) {
		T& value = input[index];
		value *= value;
	}
}

template<typename T>
__global__
void squareArray( T* input, std::size_t n ) {
	const int index = blockIdx.x*blockDim.x+threadIdx.x;
	if( index < n ) {
		T& value = input[index];
		value *= value;
	}
}

int main( int argc, char* argv[] ) {

	std::time_t start, end;
	std::time(&start);

	const std::size_t N = 100000;
	const std::size_t THREADS = 800;

	std::vector<int> hostData( N );

	ecuda::array<int> deviceData( N );
	deviceData << hostData;

	int* rawData = NULL;
	CUDA_CALL( hipMalloc( reinterpret_cast<void**>(&rawData), N*sizeof(int) ) );
	CUDA_CALL( hipMemcpy( reinterpret_cast<void*>(rawData), reinterpret_cast<const void*>(&hostData.front()), N*sizeof(int), hipMemcpyHostToDevice ) );

	dim3 grid( (N+THREADS-1)/THREADS ), threads( THREADS );
	squareArray<int><<<grid,threads>>>( deviceData );
	CUDA_CALL( hipDeviceSynchronize() );
	CUDA_CHECK_ERRORS();

	squareArray<int><<<grid,threads>>>( rawData, N );
	CUDA_CALL( hipDeviceSynchronize() );
	CUDA_CHECK_ERRORS();

	return EXIT_SUCCESS;

}
