#include "hip/hip_runtime.h"
#include <ctime>
#include <iomanip>
#include <iostream>
#include <vector>
#include "../include/ecuda/array.hpp"

template<typename T>
__device__
T doSomething( const T& value ) {
	double result = static_cast<double>(0);
	for( std::size_t i = 0; i < 10000; ++i ) {
		result += 0.0001;
//		return log(static_cast<double>(value));
	}
	return static_cast<T>(result);
}

template<typename T>
__global__
void squareArray( ecuda::array<T> input ) {
	const int index = blockIdx.x*blockDim.x+threadIdx.x;
	if( index < input.size() ) {
		T& value = input[index];
		value = doSomething(value);
		//value = log(static_cast<double>(value));
	}
}

template<typename T>
__global__
void squareArray( T* input, std::size_t n ) {
	const int index = blockIdx.x*blockDim.x+threadIdx.x;
	if( index < n ) {
		T& value = input[index];
		value = doSomething(value);
		//value = log(static_cast<double>(value));
	}
}

int main( int argc, char* argv[] ) {

	const std::size_t N = 10000000;
	const std::size_t THREADS = 800;

	std::vector<double> hostData( N );
	for( std::size_t i = 0; i < N; ++i ) hostData[i] = i+1.0;

	ecuda::array<double> deviceData( N );
	deviceData << hostData;

	double* rawData = NULL;
	CUDA_CALL( hipMalloc( reinterpret_cast<void**>(&rawData), N*sizeof(double) ) );
	CUDA_CALL( hipMemcpy( reinterpret_cast<void*>(rawData), reinterpret_cast<const void*>(&hostData.front()), N*sizeof(double), hipMemcpyHostToDevice ) );

	dim3 grid( (N+THREADS-1)/THREADS ), threads( THREADS );

//	const std::size_t ROUNDS = 100;
	{
		hipEvent_t start, stop;
		float time;
		hipEventCreate(&start);
		hipEventCreate(&stop);
//		std::time_t start, end;
//		std::time(&start);
//		for( std::size_t i = 0; i < ROUNDS; ++i ) {
		hipEventRecord( start, 0 );
		squareArray<double><<<grid,threads>>>( deviceData );
		CUDA_CALL( hipDeviceSynchronize() );
		CUDA_CHECK_ERRORS();
		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &time, start, stop );
		hipEventDestroy( start );
		hipEventDestroy( stop );
//		}
//		std::time(&end);
		std::cout << "TIME (ecuda): " << std::fixed << time << std::endl;
//		std::cout << "TIME (ecuda): " << std::fixed << difftime( end, start ) << std::endl;
		std::vector<double> results( N );
		deviceData >> results;
		for( std::size_t i = 0; i < 10; ++i ) std::cout << "[" << i << "]=" << std::fixed << results[i] << std::endl;
		std::cout << "[" << (N-1) << "]=" << std::fixed << results.back() << std::endl;
	}

	{
		hipEvent_t start, stop;
		float time;
		hipEventCreate(&start);
		hipEventCreate(&stop);
//		std::time_t start, end;
//		std::time(&start);
//		for( std::size_t i = 0; i < ROUNDS; ++i ) {
		hipEventRecord( start, 0 );
		squareArray<double><<<grid,threads>>>( rawData, N );
		CUDA_CALL( hipDeviceSynchronize() );
		CUDA_CHECK_ERRORS();
		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &time, start, stop );
		hipEventDestroy( start );
		hipEventDestroy( stop );
//		}
//		std::time(&end);
		std::cout << "TIME (raw):  " << std::fixed << time << std::endl;
//		std::cout << "TIME (raw): " << std::fixed << difftime( end, start ) << std::endl;
		std::vector<double> results( N );
		CUDA_CALL( hipMemcpy( &results.front(), rawData, N*sizeof(double), hipMemcpyDeviceToHost ) );
		for( std::size_t i = 0; i < 10; ++i ) std::cout << "[" << i << "]=" << std::fixed << results[i] << std::endl;
		std::cout << "[" << (N-1) << "]=" << std::fixed << results.back() << std::endl;
	}

	return EXIT_SUCCESS;

}
